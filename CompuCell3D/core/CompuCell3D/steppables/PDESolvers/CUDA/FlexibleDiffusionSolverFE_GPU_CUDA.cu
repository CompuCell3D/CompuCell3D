#include "hip/hip_runtime.h"
//#include "BasicUtils/BasicException.h" 
#include "FlexibleDiffusionSolverFE_GPU_CUDA.h"
#include "../DiffSecrData.h"

#include "CUDAUtilsHeader.h"
#include "../GPUSolverBasicData.h"
#include <iostream>

# define BLOCK_SIZE_FRAME (BLOCK_SIZE+2)

using std::cerr;
using std::endl;
using std::vector;
using std::string;
using std::swap;

namespace CompuCell3D {

FlexibleDiffusionSolverFE_GPU_CUDA::FlexibleDiffusionSolverFE_GPU_CUDA():h_solverParamPtr(NULL),
	d_field(NULL),
	d_celltype_field(NULL),
	d_boundary_field(NULL),
	d_scratch(NULL),
	d_solverParam(NULL),
	mem_size_field(0),
	mem_size_celltype_field(0)
{
}

FlexibleDiffusionSolverFE_GPU_CUDA::~FlexibleDiffusionSolverFE_GPU_CUDA()
{
	if (h_solverParamPtr)
		checkCudaErrors(hipHostFree(h_solverParamPtr));

	if (d_field)
		checkCudaErrors(hipFree(d_field));

	if (d_scratch)
		checkCudaErrors(hipFree(d_scratch));

	if (d_celltype_field)
		checkCudaErrors(hipFree(d_celltype_field));

	if (d_boundary_field)
		checkCudaErrors(hipFree(d_boundary_field));
		
}

void FlexibleDiffusionSolverFE_GPU_CUDA::init(int gpuDeviceIndex, LatticeType lt, size_t fieldLen){
	//hipSetDevice( /*cutGetMaxGflopsDeviceId()*/0);

	//TODO: reimplement device selector
	//not the most efficient code...
	//refactoring needed (separate device selection from user messages)
	if(gpuDeviceIndex==-1){//select the fastest GPU device
		cerr<<"Selecting the fastest GPU device...\n";
		int num_devices, device;
		hipGetDeviceCount(&num_devices);
		if (num_devices > 1) {
			  int max_multiprocessors = 0, max_device = 0;
			  for (device = 0; device < num_devices; device++) {
					  hipDeviceProp_t properties;
					  hipGetDeviceProperties(&properties, device);
					  if (max_multiprocessors < properties.multiProcessorCount) {
							  max_multiprocessors = properties.multiProcessorCount;
							  max_device = device;
					  }
			  }
			  hipDeviceProp_t properties;
			  hipGetDeviceProperties(&properties, max_device);
			  cerr<<"GPU device "<<max_device<<" selected; GPU device name: "<<properties.name<<endl;
			  hipSetDevice(max_device);
			  gpuDeviceIndex=max_device;
		}else{
			cerr<<"Only one GPU device available, will use it (#0)\n";
			hipDeviceProp_t properties;
			hipGetDeviceProperties(&properties, 0);
			cerr<<"GPU device name: "<<properties.name<<endl;
		}
	}else{
		hipError_t err=hipSetDevice(gpuDeviceIndex);
		if(err!=hipSuccess){
			cerr<<"Can't use the GPU device # "<<gpuDeviceIndex<<" (error code: "<<err<<", err message: "<<hipGetErrorString(err)<<")"<<"\n";
			exit(-1);
		}

		hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, gpuDeviceIndex);
		cerr<<"GPU device name: "<<properties.name<<endl;
	}

	alloc(fieldLen);
}

void FlexibleDiffusionSolverFE_GPU_CUDA::alloc(size_t fieldLen){
	unsigned int flags = hipHostMallocMapped;
    checkCudaErrors(hipHostAlloc((void **)&h_solverParamPtr, sizeof(SolverParams_t), flags));
	cerr<<"h_solverParamPtr-"<<h_solverParamPtr<<endl;


	// allocate device memory
    mem_size_field=fieldLen*sizeof(float);
	mem_size_celltype_field=fieldLen*sizeof(unsigned char);
    checkCudaErrors(hipMalloc((void**) &d_field, mem_size_field));

    //
    checkCudaErrors(hipMalloc((void**) &d_celltype_field, mem_size_celltype_field));

	checkCudaErrors(hipMalloc((void**) &d_boundary_field, mem_size_celltype_field));

    //
    checkCudaErrors(hipMalloc((void**) &d_scratch, mem_size_field));

	//enabling sharing of the h_solverParamPtr between host and device
    
    
	checkCudaErrors(hipHostGetDevicePointer((void **)&d_solverParam, (void *)h_solverParamPtr, 0));	
}

void FlexibleDiffusionSolverFE_GPU_CUDA::prepareSolverParams(Dim3D fieldDim, DiffusionData const &diffData){
	SolverParams_t  &h_solverParam = *h_solverParamPtr;
    h_solverParam.dimx=fieldDim.x;
    h_solverParam.dimy=fieldDim.y;
    h_solverParam.dimz=fieldDim.z;

    h_solverParam.dx=1.0;
    h_solverParam.dt=1.0;
    h_solverParam.numberOfCelltypes=2;
	
	for (int i=0 ; i<UCHAR_MAX+1 ; ++i){
		h_solverParam.diffCoef[i]=diffData.diffCoef[i];
		h_solverParam.decayCoef[i]=diffData.decayCoef[i];
		//cerr<<"h_solverParam.diffCoef["<<i<<"]="<<h_solverParam.diffCoef[i]<<endl;
	}
}

string FlexibleDiffusionSolverFE_GPU_CUDA::solverName(){
	cerr<<"Calling FlexibleDiffusionSolverFE_GPU_CUDA::solverName"<<endl;
	return "FlexibleDiffusionSolverFE_CUDA";
}

void FlexibleDiffusionSolverFE_GPU_CUDA::fieldHostToDevice(float const *h_field){
	checkCudaErrors(hipMemcpy(d_field, h_field, mem_size_field,
                              hipMemcpyHostToDevice));
}

void FlexibleDiffusionSolverFE_GPU_CUDA::fieldDeviceToHost(float *h_field)const{
	checkCudaErrors(hipMemcpy(h_field, d_scratch, mem_size_field,hipMemcpyDeviceToHost) );
}

void FlexibleDiffusionSolverFE_GPU_CUDA::swapScratchAndField(){
	swap(d_field, d_scratch);
}

void FlexibleDiffusionSolverFE_GPU_CUDA::initCellTypeArray(unsigned char *arr, size_t arrLength){
	//cerr<<"h_celltype_field->getArraySize()="<<arrLength<<" mem_size_celltype_field="<<mem_size_celltype_field<<endl;
	////h_celltype_field=cellTypeMonitorPlugin->getCellTypeArray();
	checkCudaErrors(hipMemcpy(d_celltype_field, arr, arrLength*sizeof(*d_celltype_field),hipMemcpyHostToDevice));	
}

void FlexibleDiffusionSolverFE_GPU_CUDA::initBoundaryArray(unsigned char *arr, size_t arrLength){
	checkCudaErrors(hipMemcpy(d_boundary_field, arr, arrLength*sizeof(*d_boundary_field),hipMemcpyHostToDevice));	
}



__global__ void diffSolverKernel( float* field, float* scratch,unsigned char * celltype, SolverParams_t  *solverParams){
    int bx = blockIdx.x;
    int by = blockIdx.y;
    
    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tz = threadIdx.z;

    int bz=0; //simulated blockIdx.z
    int DIMX=solverParams->dimx;
    int DIMY=solverParams->dimy;
    int DIMZ=solverParams->dimz;

    int bz_max=DIMZ/BLOCK_SIZE;

    //each thread copies data into shared memory 
    int threadsPerBlock=BLOCK_SIZE*BLOCK_SIZE*BLOCK_SIZE;

    __shared__ float fieldBlock[BLOCK_SIZE+2][BLOCK_SIZE+2][BLOCK_SIZE+2];
    __shared__ unsigned char celltypeBlock[BLOCK_SIZE+2][BLOCK_SIZE+2][BLOCK_SIZE+2];
    __shared__ float scratchBlock[BLOCK_SIZE][BLOCK_SIZE][BLOCK_SIZE];
    

    for (bz=0 ; bz<bz_max ; ++bz){


        //mapping from block,threadIdx to x,y,zof the inner frame
        int x= bx*BLOCK_SIZE+tx; 
        int y= by*BLOCK_SIZE+ty;
        int z= bz*BLOCK_SIZE+tz;        
        
        //int offset=threadsPerBlock*bx+threadsPerBlock*blockDim.x*by+DIMX*DIMY*BLOCK_SIZE*bz;
        
        fieldBlock[tx+1][ty+1][tz+1] = field[(z+1)*(DIMX+2)*(DIMY+2)+(y+1)*(DIMX+2)+x+1];
        celltypeBlock[tx+1][ty+1][tz+1] = celltype[(z+1)*(DIMX+2)*(DIMY+2)+(y+1)*(DIMX+2)+x+1];

        scratchBlock[tx][ty][tz]=0.0;  

        //fieldBlock(tx+1, ty+1, tz+1) = field[offset+tz*BLOCK_SIZE*BLOCK_SIZE+ty*BLOCK_SIZE+tx];
        if (tx==0){
            fieldBlock[0][ty+1][tz+1]=field[(z+1)*(DIMX+2)*(DIMY+2)+(y+1)*(DIMX+2)+x];
            celltypeBlock[0][ty+1][tz+1]=celltype[(z+1)*(DIMX+2)*(DIMY+2)+(y+1)*(DIMX+2)+x];
        }

        if (tx==BLOCK_SIZE-1){
            fieldBlock[BLOCK_SIZE+1][ty+1][tz+1]=field[(z+1)*(DIMX+2)*(DIMY+2)+(y+1)*(DIMX+2)+x+2];
            celltypeBlock[BLOCK_SIZE+1][ty+1][tz+1]=celltype[(z+1)*(DIMX+2)*(DIMY+2)+(y+1)*(DIMX+2)+x+2];
        }

        if (ty==0){
            fieldBlock[tx+1][0][tz+1]=field[(z+1)*(DIMX+2)*(DIMY+2)+(y)*(DIMX+2)+x+1];
            celltypeBlock[tx+1][0][tz+1]=celltype[(z+1)*(DIMX+2)*(DIMY+2)+(y)*(DIMX+2)+x+1];
        }

        if (ty==BLOCK_SIZE-1){
            fieldBlock[tx+1][BLOCK_SIZE+1][tz+1]=field[(z+1)*(DIMX+2)*(DIMY+2)+(y+2)*(DIMX+2)+x+1];
            celltypeBlock[tx+1][BLOCK_SIZE+1][tz+1]=celltype[(z+1)*(DIMX+2)*(DIMY+2)+(y+2)*(DIMX+2)+x+1];
        }

        if (tz==0){
            fieldBlock[tx+1][ty+1][0]=field[(z)*(DIMX+2)*(DIMY+2)+(y+1)*(DIMX+2)+x+1];
            celltypeBlock[tx+1][ty+1][0]=celltype[(z)*(DIMX+2)*(DIMY+2)+(y+1)*(DIMX+2)+x+1];
        }

        if (tz==BLOCK_SIZE-1){
            fieldBlock[tx+1][ty+1][BLOCK_SIZE+1]=field[(z+2)*(DIMX+2)*(DIMY+2)+(y+1)*(DIMX+2)+x+1];
            celltypeBlock[tx+1][ty+1][BLOCK_SIZE+1]=celltype[(z+2)*(DIMX+2)*(DIMY+2)+(y+1)*(DIMX+2)+x+1];
        }


        __syncthreads();

        //solve actual diff equation
        float concentrationSum =0.0;
        float dt_dx2=solverParams->dt/(solverParams->dx*solverParams->dx);

        int curentCelltype=celltypeBlock[tx+1][ty+1][tz+1];

        concentrationSum=fieldBlock[tx+2][ty+1][tz+1]+fieldBlock[tx+1][ty+2][tz+1]+fieldBlock[tx+1][ty+1][tz+2]
        +fieldBlock[tx][ty+1][tz+1]+fieldBlock[tx+1][ty][tz+1]+fieldBlock[tx+1][ty+1][tz]-6*fieldBlock[tx+1][ty+1][tz+1];

        float * diffCoef=solverParams->diffCoef;
        float * decayCoef=solverParams->decayCoef;

        
        concentrationSum*=diffCoef[curentCelltype]; 



        float varDiffSumTerm=0.0;

        //mixing central difference first derivatives with forward second derivatives does not work
        //terms due to variable diffusion coef
        ////x partial derivatives
        //varDiffSumTerm+=(diffCoef[celltypeBlock[tx+2][ty+1][tz+1]]-diffCoef[celltypeBlock[tx][ty+1][tz+1]])*(fieldBlock[tx+2][ty+1][tz+1]-fieldBlock[tx][ty+1][tz+1]);
        ////y partial derivatives
        //varDiffSumTerm+=(diffCoef[celltypeBlock[tx+1][ty+2][tz+1]]-diffCoef[celltypeBlock[tx+1][ty][tz+1]])*(fieldBlock[tx+1][ty+2][tz+1]-fieldBlock[tx+1][ty][tz+1]);
        ////z partial derivatives
        //varDiffSumTerm+=(diffCoef[celltypeBlock[tx+1][ty+1][tz+2]]-diffCoef[celltypeBlock[tx+1][ty+1][tz]])*(fieldBlock[tx+1][ty+1][tz+2]-fieldBlock[tx+1][ty+1][tz]);        

        //scratchBlock[tx][ty][tz]=diffConst*(concentrationSum-6*fieldBlock[tx+1][ty+1][tz+1])+fieldBlock[tx+1][ty+1][tz+1];

        //scratchBlock[tx][ty][tz]=dt_4dx2*(concentrationSum+4*varDiffSumTerm)+fieldBlock[tx+1][ty+1][tz+1];


        //scratchBlock[tx][ty][tz]=dt_4dx2*(concentrationSum+varDiffSumTerm)+fieldBlock[tx+1][ty+1][tz+1];


        //using forward first derivatives
        //x partial derivatives
        varDiffSumTerm+=(diffCoef[celltypeBlock[tx+2][ty+1][tz+1]]-diffCoef[curentCelltype])*(fieldBlock[tx+2][ty+1][tz+1]-fieldBlock[tx+1][ty+1][tz+1]);
        //y partial derivatives
        varDiffSumTerm+=(diffCoef[celltypeBlock[tx+1][ty+2][tz+1]]-diffCoef[curentCelltype])*(fieldBlock[tx+1][ty+2][tz+1]-fieldBlock[tx+1][ty+1][tz+1]);
        //z partial derivatives
        varDiffSumTerm+=(diffCoef[celltypeBlock[tx+1][ty+1][tz+2]]-diffCoef[curentCelltype])*(fieldBlock[tx+1][ty+1][tz+2]-fieldBlock[tx+1][ty+1][tz+1]);        

         
        //OK
        scratchBlock[tx][ty][tz]=dt_dx2*(concentrationSum+varDiffSumTerm)+(1-solverParams->dt*decayCoef[curentCelltype])*fieldBlock[tx+1][ty+1][tz+1];

    
        
        //simple consistency check
        //scratchBlock[tx][ty][tz]=concentrationSum;
        //scratchBlock[tx][ty][tz]=fieldBlock[tx+2][ty+1][tz+1]+fieldBlock[tx][ty+1][tz+1]+fieldBlock[tx+1][ty+2][tz+1]+fieldBlock[tx+1][ty][tz+1]+fieldBlock[tx+1][ty+1][tz+2]+fieldBlock[tx+1][ty+1][tz];

        //scratchBlock[tx][ty][tz]=fieldBlock[tx+1][ty+1][tz+1];

        //fieldBlock[tx+1][ty+1][tz+1]=3000.0f;
        __syncthreads();

        //copy scratchBlock to scratch field on the device

        scratch[(z+1)*(DIMX+2)*(DIMY+2)+(y+1)*(DIMX+2)+x+1]=scratchBlock[tx][ty][tz];
        //scratch[(z+1)*(DIMX+2)*(DIMY+2)+(y+1)*(DIMX+2)+x+1]=3000.0;

        __syncthreads();

        //boundary condition
        //if(x==0){
        //    scratch[(z+1)*(DIMX+2)*(DIMY+2)+(y+1)*(DIMX+2)+x]=scratch[(z+1)*(DIMX+2)*(DIMY+2)+(y+1)*(DIMX+2)+x+1];
        //}

        //if(x==solverParams->dimx-1){
        //    scratch[(z+1)*(DIMX+2)*(DIMY+2)+(y+1)*(DIMX+2)+x+2]=scratch[(z+1)*(DIMX+2)*(DIMY+2)+(y+1)*(DIMX+2)+x+1];
        //}

        //if(y==0){
        //    scratch[(z+1)*(DIMX+2)*(DIMY+2)+(y)*(DIMX+2)+x+1]=scratch[(z+1)*(DIMX+2)*(DIMY+2)+(y+1)*(DIMX+2)+x+1];
        //}

        //if(y==solverParams->dimy-1){
        //    scratch[(z+1)*(DIMX+2)*(DIMY+2)+(y+2)*(DIMX+2)+x+1]=scratch[(z+1)*(DIMX+2)*(DIMY+2)+(y+1)*(DIMX+2)+x+1];
        //}

        //if(z==0){
        //    scratch[(z)*(DIMX+2)*(DIMY+2)+(y+1)*(DIMX+2)+x+1]=scratch[(z+1)*(DIMX+2)*(DIMY+2)+(y+1)*(DIMX+2)+x+1];
        //}

        //if(z==solverParams->dimz-1){
        //    scratch[(z+2)*(DIMX+2)*(DIMY+2)+(y+1)*(DIMX+2)+x+1]=scratch[(z+1)*(DIMX+2)*(DIMY+2)+(y+1)*(DIMX+2)+x+1];
        //}

        
    }
    //__syncthreads();
}

void FlexibleDiffusionSolverFE_GPU_CUDA::diffuseSingleField(){
	//we cannot access device variable (e.g. d_solverParam) from this part of the code - only kernel is allowed to do this
    //here we are using page-locked memory to share SolverParams_t structure between device and host
    unsigned int dimX=h_solverParamPtr->dimx;
    unsigned int dimY=h_solverParamPtr->dimy;
    unsigned int dimZ=h_solverParamPtr->dimz;

    SolverParams_t * d_solverParamFromMappedMemory;
    hipHostGetDevicePointer((void **)&d_solverParamFromMappedMemory, (void *)h_solverParamPtr, 0);

    //cutilSafeCall(hipMemcpy(d_solverParamFromMappedMemory, h_solverParam, sizeof(SolverParams_t ),hipMemcpyHostToDevice) );

    // setup execution parameters
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(dimX / threads.x, dimY / threads.y);

    diffSolverKernel<<< grid, threads >>>(d_field, d_scratch,d_celltype_field,d_solverParamFromMappedMemory);
    //diffSolverKernel<<< grid, threads >>>(d_field, d_scratch,d_celltype_field,d_solverParam);
    hipDeviceSynchronize();//TODO: this synchronization looks redundant. Copying memory back to host implies implicit synchronization
}



}//namespace CompuCell3D